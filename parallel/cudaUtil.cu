#include "cudaUtil.h"
#include <hip/hip_runtime.h>

/* Function: AllocateDevice 
 *--------------------------
 * Allocate memory on device 
 * 
 * size_t size: size of memory to allocate 
 *
 * returns a void * pointer to allocated memory on device
 */
void* AllocateDevice(size_t size)
{
  void* ret;
  hipMalloc(&ret, size);
  return ret;
}

/* Function: CudaMemCpyToSymbol 
 *------------------------------
 * Copy memory into device cache 
 * 
 * void *D_device: pointer to allocated location in device memory
 * void *D_host: pointer to allocated memory in host to copy data from
 *
 * returns nothing
 */
void CudaMemCpyToSymbol(void *D_device, void *D_host, size_t size)
{
  hipMemcpyToSymbol(HIP_SYMBOL(D_device), D_host, size);
}

/* Function: CopyToDevice 
 *------------------------
 * Copy memory from host to device
 *
 * void *D_device: pointer to allocated location in device memory to copy data into
 * void *D_host: pointer to allocated memory in host to copy data from
 * size_t size: size of memory to copy over 
 * 
 * returns nothing
 */
void CopyToDevice(void* D_device, void* D_host, size_t size)
{
  hipMemcpy(D_device, D_host, size, hipMemcpyHostToDevice);
}

/* Function: CopyFromDevice
 *--------------------------
 * Copy memory from device to host 
 *
 * void *D_host: pointer to allocated location in host memory to copy data into
 * void *D_device: pointer to allocated memory in device to copy data from
 * size_t size: size of memory to copy over
 *
 * returns nothing
 */
void CopyFromDevice(void* D_host, void* D_device, size_t size)
{
  hipMemcpy(D_host, D_device, size, hipMemcpyDeviceToHost);
}

/* Function: FreeDevice 
 *---------------------- 
 * Free allocated memory on device 
 * 
 * void *D_device: pointer to allocated memory in device to free
 *
 * returns nothing
 */
void FreeDevice(void* D_device)
{
  hipFree(D_device);
}
