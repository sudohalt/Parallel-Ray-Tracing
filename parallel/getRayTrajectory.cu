#include "hip/hip_runtime.h"
/* Parallel code to calculate ray trace trajectory */
#include <hip/hip_runtime.h>
#include "cudaUtil.h"
#include "getRayTrajectory_kernel.h"

int main(int argc, char *argv[])
{
	/* Declare doubles */
	double x_min = 0.0;
	double x_max = 0.0;
	double y_min = 0.0;
	double y_max - 0.0;
	double delta = 0.0;
	double dt = 0.0;
	double y0 = 0.0; 
	double theta0 = 0.0; 

	/* Allocate space for host arrays */
	double** n_profile = malloc(sizeof(double) * /* Some size */);
	double** Dx = malloc(sizeof(double) * /* Some size */);
	double** Dy = malloc(sizeof(double) * /* Some size */);
	double** Dxx = malloc(sizeof(double) * /* Some size */);
	double** Dxy = malloc(sizeof(double) * /* Some size */);
	double** Dyx = malloc(sizeof(double) * /* Some size */);
	double** Dyy = malloc(sizeof(double) * /* Some size */);
	
	/* Fill host arrays with necessary data */

	/* Allocate space for device arrays */	
	double** n_profileDevice = AllocateDevice(/* Some size */);
	double** DxDevice = AllocateDevice(/* Some size */);
	double** DyDevice = AllocateDevice(/* Some size */);
	double** DxxDevice = AllocateDevice(/* Some size */);
	double** DxyDevice = AllocateDevice(/* Some size */);
	double** DyxDevice = AllocateDevice(/* Some size */);
	double** DyyDevice = AllocateDevice(/* Some size */);

	/* Copy host array contents to device array */	
	CopyToDevice(n_profileDevice, n_profile, /* Some size */); 
	CopyToDevice(DxDevice, Dx, /* Some size */);
	CopyToDevice(DyDevice, Dy, /* Some size */);
	CopyToDevice(DxxDevice, Dxx, /* Some size */);
	CopyToDevice(DxyDevice, Dxy, /* Some size */);
	CopyToDevice(DyxDevice, Dyx, /* Some size */);
	CopyToDevice(DyyDevice, Dyy, /* Some size */);

	/* Set up dimensions and threads */
	dim3 gridDimensions;
	gridDimensions.x = 0;
	gridDimensions.y = 0;
	gridDimensions.z = 0;
	dim3 blockDimensions;
	blockDimensions.x = 0;
	blockDimensions.y = 0;
	blockDimensions.z = 0;

	/* Call kernel function */
	getRayTrajectory_kernel<<gridDimensions, blockDimensions>>(n_profileDevice, DxDevice, 
			DyDevice, DxxDevice, DxyDevice, DyxDevice, DyyDevice, x_min, x_max, 
			y_min, y_max, delta, dt, y0, theta0);
	
	/* Copy contents from device arrays to host arrays */

	/* Free devcice memory */
	FreeDevice(n_profileDevice);
	FreeDevice(DxDevice);
	FreeDevice(DyDevice);
	FreeDevice(DxxDevice);
	FreeDevice(DxyDevice);
	FreeDevice(DyxDevice);
	FreeDevice(DyyDevice);

	/* Free host memory */
	free(n_profile);
	free(Dx);
	free(Dy);
	free(Dxx);
	free(Dxy);
	free(Dyx);
	free(Dyy);

	return 0;
}
