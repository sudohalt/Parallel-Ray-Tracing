#include "hip/hip_runtime.h"
/* Parallel code to calculate ray trace trajectory */
#include <hip/hip_runtime.h>
#include "getRayTrajectory_kernel.h"

int getRayTrajectory(double** n_profile, double** Dx,double** Dy, double** Dxx, double** Dxy, 
		     double* Dyx, double* Dyy, double x_min, double x_max, double y_min, 
		     double y_max, double delta, double dt, double y0, double theta0)
{
	/* Do parallel stuff */
}
